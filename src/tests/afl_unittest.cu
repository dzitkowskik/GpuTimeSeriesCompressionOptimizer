#include "afl_unittest.hpp"
#include "helpers/helper_comparison.cuh"

namespace ddj
{

INSTANTIATE_TEST_CASE_P(
    RandomFloatNumbersCompression_Afl_Inst,
    AflCompressionTest,
    ::testing::Values(10, 20));

TEST_P(AflCompressionTest, CompressionOfRandomInt_size)
{
    int real_size = GetParam();
    int compressed_size;
    int decompressed_size;
    AFLCompressionMetadata metadata;
    void* compressedData = compression.Encode(
        d_random_data,
        real_size,
        compressed_size,
        metadata);

    void* decompressedData = compression.Decode(
        (int*)compressedData,
        compressed_size,
        decompressed_size,
        metadata);

    EXPECT_EQ(real_size, decompressed_size);

    CUDA_CALL(hipFree(compressedData));
    CUDA_CALL(hipFree(decompressedData));
}

TEST_P(AflCompressionTest, CompressionOfRandomInt_data)
{
    int real_size = GetParam();
    int compressed_size;
    int decompressed_size;
    AFLCompressionMetadata metadata;
    void* compressedData = compression.Encode(
        d_random_data,
        real_size,
        compressed_size,
        metadata);

    int* decompressedData = (int*)compression.Decode(
        (int*)compressedData,
        compressed_size,
        decompressed_size,
        metadata);

    EXPECT_TRUE(CompareDeviceArrays(d_random_data, decompressedData, real_size));

    CUDA_CALL(hipFree(compressedData));
    CUDA_CALL(hipFree(decompressedData));
}

} /* namespace ddj */
