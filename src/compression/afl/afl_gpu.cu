#include "hip/hip_runtime.h"
#include "afl_gpu.cuh"
#include "core/cuda_macros.cuh"
#include "core/macros.h"
#include <stdio.h>

template < typename T, char CWARP_SIZE >
__host__ void run_afl_compress_cpu( const unsigned int bit_length, T *data, T *compressed_data, const unsigned long length, unsigned long comprLength)
{

    const unsigned int block_size = CWARP_SIZE * 8;
    const unsigned long block_number = (length + block_size * CWORD_SIZE(T) - 1) / (block_size * CWORD_SIZE(T));

    unsigned int tid, bid;

    for (tid = 0, bid = 0; bid <= block_number; tid++)
    {
        if (tid == block_size)
        {
           tid = 0;
           bid += 1;
        }

        unsigned int warp_lane = (tid % CWARP_SIZE);
        unsigned long data_block = bid * block_size + tid - warp_lane;
        unsigned long data_id = data_block * CWORD_SIZE(T) + warp_lane;
        unsigned long cdata_id = data_block * bit_length + warp_lane;

        afl_compress_base_gpu <T, CWARP_SIZE> (bit_length, data_id, cdata_id, data, compressed_data, length, comprLength);
    }
}

template < typename T, char CWARP_SIZE >
__host__ void run_afl_compress_value_cpu( const unsigned int bit_length, T *data, T *compressed_data, const unsigned long length)
{

    unsigned long tid;

    for (tid = 0; tid < length; tid++)
        afl_compress_base_value_gpu <T, CWARP_SIZE> (bit_length, compressed_data, tid, data[tid]);
}

template < typename T, char CWARP_SIZE >
__host__ void run_afl_decompress_cpu(const unsigned int bit_length, T *compressed_data, T *decompress_data, unsigned long length)
{
    const unsigned int block_size = CWARP_SIZE * 8;
    const unsigned long block_number = (length + block_size * CWORD_SIZE(T) - 1) / (block_size * CWORD_SIZE(T));

    unsigned long tid, bid;

    for (tid = 0, bid = 0; bid < block_number; tid++)
    {
        if (tid == block_size)
        {
           tid = 0;
           bid += 1;
        }

        unsigned int warp_lane = (tid % CWARP_SIZE);
        unsigned long data_block = bid * block_size + tid - warp_lane;
        unsigned long data_id = data_block * CWORD_SIZE(T) + warp_lane;
        unsigned long cdata_id = data_block * bit_length + warp_lane;

        afl_decompress_base_gpu <T, CWARP_SIZE> (bit_length, cdata_id, data_id, compressed_data, decompress_data, length);
    }
}

template < typename T, char CWARP_SIZE >
__host__ void run_afl_compress_gpu(const unsigned int bit_length, T *data, T *compressed_data, unsigned long length, unsigned long comprLength)
{
    const unsigned int block_size = CWARP_SIZE * 8; // better occupancy
    const unsigned long block_number = (length + block_size * CWORD_SIZE(T) - 1) / (block_size * CWORD_SIZE(T));
    afl_compress_gpu <T, CWARP_SIZE> <<<block_number, block_size>>> (bit_length, data, compressed_data, length, comprLength);
}

template < typename T, char CWARP_SIZE >
__host__ void run_afl_decompress_gpu(const unsigned int bit_length, T *compressed_data, T *data, unsigned long length)
{
    const unsigned int block_size = CWARP_SIZE * 8; // better occupancy
    const unsigned long block_number = (length + block_size * CWORD_SIZE(T) - 1) / (block_size * CWORD_SIZE(T));
    afl_decompress_gpu <T, CWARP_SIZE> <<<block_number, block_size>>> (bit_length, compressed_data, data, length);
}

template < typename T, char CWARP_SIZE >
__host__ void run_afl_decompress_value_gpu(const unsigned int bit_length, T *compressed_data, T *data, unsigned long length)
{
    const unsigned int block_size = CWARP_SIZE * 8; // better occupancy
    const unsigned long block_number = (length + block_size * CWORD_SIZE(T) - 1) / (block_size);
    afl_decompress_value_gpu <T, CWARP_SIZE> <<<block_number, block_size>>> (bit_length, compressed_data, data, length);
}

template < typename T, char CWARP_SIZE >
__global__ void afl_compress_gpu (const unsigned int bit_length, T *data, T *compressed_data, unsigned long length, unsigned long comprLength)
{
    const unsigned int warp_lane = (threadIdx.x % CWARP_SIZE);
    const unsigned long data_block = blockIdx.x * blockDim.x + threadIdx.x - warp_lane;
    const unsigned long data_id = data_block * CWORD_SIZE(T) + warp_lane;
    const unsigned long cdata_id = data_block * bit_length + warp_lane;

    afl_compress_base_gpu <T, CWARP_SIZE> (bit_length, data_id, cdata_id, data, compressed_data, length, comprLength);
}

template < typename T, char CWARP_SIZE >
__global__ void afl_decompress_gpu (const unsigned int bit_length, T *compressed_data, T * decompress_data, unsigned long length)
{
    const unsigned int warp_lane = (threadIdx.x % CWARP_SIZE);
    const unsigned long data_block = blockIdx.x * blockDim.x + threadIdx.x - warp_lane;
    const unsigned long data_id = data_block * CWORD_SIZE(T) + warp_lane;
    const unsigned long cdata_id = data_block * bit_length + warp_lane;

    afl_decompress_base_gpu <T, CWARP_SIZE> (bit_length, cdata_id, data_id, compressed_data, decompress_data, length);
}

template < typename T, char CWARP_SIZE >
__global__ void afl_decompress_value_gpu (const unsigned int bit_length, T *compressed_data, T * decompress_data, unsigned long length)
{
    const unsigned long tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < length)
    {
        decompress_data[tid] = afl_decompress_base_value_gpu <T, CWARP_SIZE> (bit_length, compressed_data, tid);
    }
}


template <typename T, char CWARP_SIZE>
__device__  __host__ void afl_compress_base_gpu (
		const unsigned int bit_length,
		unsigned long data_id,
		unsigned long comp_data_id,
		T *data,
		T *compressed_data,
		unsigned long length,
		unsigned long comprLength)
{
    T v1, value = 0;
    unsigned int v1_pos=0, v1_len;
    unsigned long pos=comp_data_id, pos_data=data_id;

    for (unsigned int i = 0; (i < CWORD_SIZE(T)) && (pos_data < length); ++i)
    {
        v1 = data[pos_data];
        pos_data += CWARP_SIZE;

        if (v1_pos >= CWORD_SIZE(T) - bit_length && (pos < comprLength)){
            v1_len = CWORD_SIZE(T) - v1_pos;
            value = value | (GETNBITS(v1, v1_len) << v1_pos);

            compressed_data[pos] = value;

            v1_pos = bit_length - v1_len;
            value = GETNPBITS(v1, v1_pos, v1_len);

            pos += CWARP_SIZE;
        } else {
            v1_len = bit_length;
            value = value | (GETNBITS(v1, v1_len) << v1_pos);
            v1_pos += v1_len;
        }
    }
    if ((pos_data >= length)  && (pos_data < length + CWARP_SIZE) && (pos < comprLength))
    {
        compressed_data[pos] = value;
    }
}

template <typename T, char CWARP_SIZE>
__device__ __host__ void afl_decompress_base_gpu (const unsigned int bit_length, unsigned long comp_data_id, unsigned long data_id, T *compressed_data, T *data, unsigned long length)
{
    unsigned long pos = comp_data_id, pos_decomp = data_id;
    unsigned int v1_pos = 0, v1_len;
    T v1, ret;

    if (pos_decomp > length ) // Decompress not more elements then length
        return;
    v1 = compressed_data[pos];
    for (unsigned int i = 0; i < CWORD_SIZE(T) && pos_decomp < length; ++i)
    {
        if (v1_pos > CWORD_SIZE(T) - bit_length){
            v1_len = CWORD_SIZE(T) - v1_pos;
            ret = GETNPBITS(v1, v1_len, v1_pos);

            pos += CWARP_SIZE;
            v1 = compressed_data[pos];

            v1_pos = bit_length - v1_len;
            ret = ret | ((GETNBITS(v1, v1_pos))<< v1_len);
        } else {
            v1_len = bit_length;
            ret = GETNPBITS(v1, v1_len, v1_pos);
            v1_pos += v1_len;
        }

        data[pos_decomp] = ret;
        pos_decomp += CWARP_SIZE;
    }
}

template <typename T, char CWARP_SIZE>
__device__ __host__ T afl_decompress_base_value_gpu (
        const unsigned int bit_length,
        T *compressed_data,
        unsigned long pos
        )
{
    const unsigned int data_block = pos / (CWARP_SIZE * CWORD_SIZE(T));
    const unsigned int pos_in_block = (pos % (CWARP_SIZE * CWORD_SIZE(T)));
    const unsigned int pos_in_warp_lane = pos_in_block % CWARP_SIZE;
    const unsigned int pos_in_warp_comp_block = pos_in_block / CWARP_SIZE;

    const unsigned long cblock_id =
        data_block * ( CWARP_SIZE * bit_length)
        + pos_in_warp_lane
        + ((pos_in_warp_comp_block * bit_length) / CWORD_SIZE(T)) * CWARP_SIZE;

    const unsigned int bit_pos = pos_in_warp_comp_block * bit_length % CWORD_SIZE(T);
    const unsigned int bit_ret = bit_pos <= CWORD_SIZE(T)  - bit_length  ? bit_length : CWORD_SIZE(T) - bit_pos;

    T ret = GETNPBITS(compressed_data[cblock_id], bit_ret, bit_pos);

    if (bit_ret < bit_length)
        ret |= GETNBITS(compressed_data[cblock_id+CWARP_SIZE], bit_length - bit_ret) << bit_ret;

    return ret;
}

template <typename T, char CWARP_SIZE>
__device__ __host__ void afl_compress_base_value_gpu (
        const unsigned int bit_length,
        T *compressed_data,
        unsigned long pos,
        T value
        )
{
    const unsigned int data_block = pos / (CWARP_SIZE * CWORD_SIZE(T));
    const unsigned int pos_in_block = (pos % (CWARP_SIZE * CWORD_SIZE(T)));
    const unsigned int pos_in_warp_lane = pos_in_block % CWARP_SIZE;
    const unsigned int pos_in_warp_comp_block = pos_in_block / CWARP_SIZE;

    const unsigned long cblock_id =
        data_block * ( CWARP_SIZE * bit_length) // move to data block
        + pos_in_warp_lane // move to starting position in data block
        + ((pos_in_warp_comp_block * bit_length) / CWORD_SIZE(T)) * CWARP_SIZE; // move to value

    const unsigned int bit_pos = pos_in_warp_comp_block * bit_length % CWORD_SIZE(T);
    const unsigned int bit_ret = bit_pos <= CWORD_SIZE(T)  - bit_length  ? bit_length : CWORD_SIZE(T) - bit_pos;


    SETNPBITS(compressed_data + cblock_id, value, bit_ret, bit_pos);

    if (bit_ret < bit_length)
        SETNPBITS(compressed_data + cblock_id + CWARP_SIZE, (T)(value >> bit_ret), bit_length - bit_ret, 0);
}

// For now only those versions are available and will be compiled and linked
// This is intentional !!
#define GFL_SPEC(X, A) \
    template __host__ void run_afl_compress_gpu <X, A> (const unsigned int bit_length, X *data, X *compressed_data, unsigned long length, unsigned long);\
    template __host__ void run_afl_decompress_gpu <X, A> (const unsigned int bit_length, X *data, X *compressed_data, unsigned long length);\
    template __host__ void run_afl_compress_cpu <X, A> (const unsigned int bit_length, X *data, X *compressed_data, unsigned long length, unsigned long);\
    template __host__ void run_afl_decompress_cpu <X, A> (const unsigned int bit_length, X *data, X *compressed_data, unsigned long length);\
    template __host__ void run_afl_compress_value_cpu <X, A> (const unsigned int bit_length, X *data, X *compressed_data, unsigned long length);\
    template __host__ void run_afl_decompress_value_gpu <X, A> (const unsigned int bit_length, X *compressed_data, X *data, unsigned long length);

// A fast aligned version WARP_SIZE = 32
#define AFL_SPEC(X) GFL_SPEC(X, 32)
FOR_EACH(AFL_SPEC, char, short, int, long, unsigned int, unsigned long)

// Non aligned version - identical to classical CPU/GPU version (up to 10x slower then AFL)
#define FL_SPEC(X) GFL_SPEC(X, 1)
FOR_EACH(FL_SPEC, char, short, int, long, unsigned int, unsigned long)
