#include "hip/hip_runtime.h"
/*
 *  float_encoding.cu
 *
 *  Created on: 30 paź 2015
 *      Author: Karol Dzitkowski
 */

#include <compression/float/float_encoding.hpp>
#include <util/transform/cuda_array_transform.hpp>
#include <util/statistics/cuda_array_statistics.hpp>
#include "core/macros.h"
#include <limits>
#include <cmath>

namespace ddj
{
	template<typename T>
	SharedCudaPtrVector<char> FloatEncoding::Encode(SharedCudaPtr<T> data)
	{
		CUDA_ASSERT_RETURN( hipGetLastError() );
	    LOG4CPLUS_INFO_FMT(_logger, "FLOAT encoding START: data size = %lu", data->size());

		if(data->size() <= 0)
			return SharedCudaPtrVector<char>{
						CudaPtr<char>::make_shared(),
						CudaPtr<char>::make_shared()
						};

		int precision = CudaArrayStatistics().Precision(data);
		SharedCudaPtr<int> resultData;
		FloatingPointToIntegerOperator<T, int> op { precision };

		// Make sure we won't overflow
		bool transform = false;
		if(precision < MAX_PRECISION)
		{
			auto minMax = CudaArrayStatistics().MinMax(data);
			int scaleFactor = std::pow(10, precision);
			if((std::get<0>(minMax) * scaleFactor) > std::numeric_limits<int>::min() &&
					(std::get<1>(minMax) * scaleFactor) < std::numeric_limits<int>::max())
				transform = true;
			else precision = MAX_PRECISION;
		}

		if(transform)
			resultData = CudaArrayTransform().Transform<T, int>(data, op);
		else
			resultData = CastSharedCudaPtr<T, int>(data->copy());

		auto resultMetadata = CudaPtr<char>::make_shared(sizeof(int));
		resultMetadata->fillFromHost((char*)&precision, sizeof(int));

		CUDA_ASSERT_RETURN( hipGetLastError() );
	    LOG4CPLUS_INFO(_logger, "FLOAT enoding END");

		return SharedCudaPtrVector<char> { resultMetadata, MoveSharedCudaPtr<int, char>(resultData) };
	}

	template<typename T>
	SharedCudaPtr<T> FloatEncoding::Decode(SharedCudaPtrVector<char> input)
	{
		LOG4CPLUS_INFO_FMT(
			_logger,
			"FLOAT decoding START: input[0] size = %lu, input[1] size = %lu",
			input[0]->size(), input[1]->size()
		);

		if(input[1]->size() <= 0)
			return CudaPtr<T>::make_shared();

		auto metadata = input[0];
		auto data = MoveSharedCudaPtr<char, int>(input[1]);

		int precision;
		CUDA_CALL( hipMemcpy(&precision, metadata->get(), sizeof(int), CPY_DTH) );

		SharedCudaPtr<T> result;
		IntegerToFloatingPointOperator<int, T> op { precision };
		if(precision < MAX_PRECISION)
			result = CudaArrayTransform().Transform<int, T>(data, op);
		else
			result = CastSharedCudaPtr<int, T>(data->copy());

		CUDA_ASSERT_RETURN( hipGetLastError() );
	    LOG4CPLUS_INFO(_logger, "FLOAT decoding END");

		return result;
	}

#define FLOAT_ENCODING_SPEC(X) \
	template SharedCudaPtrVector<char> FloatEncoding::Encode<X>(SharedCudaPtr<X> data); \
	template SharedCudaPtr<X> FloatEncoding::Decode<X>(SharedCudaPtrVector<char> data);
FOR_EACH(FLOAT_ENCODING_SPEC, char, short, double, float, int, long, long long, unsigned int)

} /* namespace ddj */
