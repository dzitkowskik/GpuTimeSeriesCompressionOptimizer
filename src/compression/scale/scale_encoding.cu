#include "hip/hip_runtime.h"
/*
 * scale_encoding.cpp
 *
 *  Created on: 24-04-2015
 *      Author: Karol Dzitkowski
 */

#include "scale_encoding.hpp"
#include "core/macros.h"
#include "core/cuda_macros.cuh"
#include "core/cuda_launcher.cuh"

#include <hip/hip_runtime_api.h>
#include <thrust/extrema.h>
#include <thrust/device_ptr.h>

namespace ddj {

template<typename T>
__global__ void scaleEncodeKernel(T* data, int size, T* result_data, T min, T* result_metadata)
{
	unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx >= size) return;
	result_data[idx] = data[idx] - min;
	result_metadata[0] = min;
}

template<typename T>
SharedCudaPtrVector<char> ScaleEncoding::Encode(SharedCudaPtr<T> data)
{
	if(data->size() <= 0)
		return SharedCudaPtrVector<char>{ CudaPtr<char>::make_shared(), CudaPtr<char>::make_shared() };

	// ALLOCATE RESULTS
	auto result_data = CudaPtr<char>::make_shared(data->size()*sizeof(T));
	auto result_metadata = CudaPtr<char>::make_shared(sizeof(T));

	// GET MIN VALUE OF DATA
	thrust::device_ptr<T> data_ptr(data->get());
	T min = thrust::min_element(data_ptr, data_ptr+data->size())[0];

	// SCALE DATA BY MIN VALUE
	this->_policy.setSize(data->size());
	hipLaunchByPtr(this->_policy, scaleEncodeKernel<T>,
			data->get(),
			data->size(),
			(T*)result_data->get(),
			min,
			(T*)result_metadata->get());

	hipDeviceSynchronize();

	return SharedCudaPtrVector<char> {result_metadata, result_data};
}

template<typename T>
__global__ void scaleDecodeKernel(T* data, int size, T* result, T* min)
{
	unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx >= size) return;
	result[idx] = data[idx] + *min;
}

template<typename T>
SharedCudaPtr<T> ScaleEncoding::Decode(SharedCudaPtrVector<char> input)
{
	if(input[1]->size() <= 0)
		return CudaPtr<T>::make_shared();

	auto metadata = input[0];
	auto data = input[1];

	int size = data->size()/sizeof(T);
	auto result = CudaPtr<T>::make_shared(size);

	this->_policy.setSize(size);
	hipLaunchByPtr(this->_policy, scaleDecodeKernel<T>,
			(T*)data->get(),
			size,
			result->get(),
			(T*)metadata->get());

	hipDeviceSynchronize();

	return result;
}

#define SCALE_SPEC(X) \
	template SharedCudaPtrVector<char> ScaleEncoding::Encode<X>(SharedCudaPtr<X> data); \
	template SharedCudaPtr<X> ScaleEncoding::Decode<X>(SharedCudaPtrVector<char> data);
FOR_EACH(SCALE_SPEC, char, short, double, float, int, long, long long, unsigned int)

} /* namespace ddj */
