#include "hip/hip_runtime.h"
/*
 * delta_encoding.cu
 *
 *  Created on: 18-04-2015
 *      Author: Karol Dzitkowski
 */

#include "delta_encoding.cuh"
#include "helpers/helper_macros.h"
#include <hip/hip_runtime_api.h>
#include <thrust/device_ptr.h>

#define DELTA_ENCODING_GPU_BLOCK_SIZE 64
#define DELTA_DECODING_GPU_BLOCK_SIZE 64

namespace ddj
{

template<typename T>
__global__ void deltaEncodeKernel(T* data, int size, T* result)
{
	unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx+1 >= size) return;
	register T v1 = data[idx];
	register T v2 = data[idx+1];
	result[idx] = v2 - v1;
}

template<typename T>
SharedCudaPtr<char> DeltaEncoding::Encode(SharedCudaPtr<T> data)
{
	int block_size = DELTA_ENCODING_GPU_BLOCK_SIZE;
	int block_cnt = (data->size() + block_size - 1) / block_size;

	auto result = CudaPtr<char>::make_shared(data->size()*sizeof(T));
	deltaEncodeKernel<T><<<block_size, block_cnt>>>(
			data->get(),
			data->size(),
			(T*)(result->get()+sizeof(T)));

	CUDA_CALL( hipMemcpy(result->get(), data->get(), sizeof(T), hipMemcpyDeviceToDevice) );
	hipDeviceSynchronize();

	return result;
}

template<typename T>
__global__ void deltaDecodeKernel(T* data, int size, T first, T* result)
{
	unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx >= size) return;
	for(int i=0; i<idx; i++)
		first += data[i];
	result[idx] = first;
}

template<typename T>
SharedCudaPtr<T> DeltaEncoding::Decode(SharedCudaPtr<char> data)
{
	int size = data->size()/sizeof(T);
	int block_size = DELTA_DECODING_GPU_BLOCK_SIZE;
	int block_cnt = (size + block_size - 1) / block_size;

	thrust::device_ptr<T> data_ptr((T*)data->get());
	auto first = data_ptr[0];

	auto result = CudaPtr<T>::make_shared(size);
	deltaDecodeKernel<T><<<block_size, block_cnt>>>(
			(T*)(data->get()+sizeof(T)), size, first, result->get());
	hipDeviceSynchronize();

	return result;
}

#define SCALE_SPEC(X) \
	template SharedCudaPtr<char> DeltaEncoding::Encode<X>(SharedCudaPtr<X> data); \
	template SharedCudaPtr<X> DeltaEncoding::Decode<X>(SharedCudaPtr<char> data);
FOR_EACH(SCALE_SPEC, double, float, int, long, long long, unsigned int, unsigned long, unsigned long long)


} /* namespace ddj */
