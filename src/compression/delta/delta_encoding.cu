#include "hip/hip_runtime.h"
/*
 *  delta_encoding.cu
 *
 *  Created on: 18-04-2015
 *      Author: Karol Dzitkowski
 */

#include "delta_encoding.hpp"
#include "helpers/helper_macros.h"
#include "helpers/helper_cuda.cuh"

#include <hip/hip_runtime_api.h>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>

namespace ddj {

template<typename T>
__global__ void deltaEncodeKernel(T* data, int size, T* result)
{
	unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx+1 >= size) return;
	register T v1 = data[idx];
	register T v2 = data[idx+1];
	result[idx] = v2 - v1;
}

template<typename T>
SharedCudaPtrVector<char> DeltaEncoding::Encode(SharedCudaPtr<T> data)
{
	// MAKE DELTA ENCODING
	auto result_data = CudaPtr<char>::make_shared((data->size() - 1) * sizeof(T));
	this->_policy.setSize(data->size());
	hipLaunchByPtr(this->_policy, deltaEncodeKernel<T>,
		data->get(),
		data->size(),
		(T*)(result_data->get())
	);

	// SAVE FIRST VALUE TO METADATA
	auto result_metadata = CudaPtr<char>::make_shared(sizeof(T));
	CUDA_CALL( hipMemcpy(result_metadata->get(), data->get(), sizeof(T), hipMemcpyDeviceToDevice) );

	hipDeviceSynchronize();

	return SharedCudaPtrVector<char> {result_metadata, result_data};
}

template<typename T>
__global__ void addValueKernel(T* data, const int size, T* value)
{
	unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx >= size) return;
	data[idx] += *value;
}

template<typename T>
SharedCudaPtr<T> DeltaEncoding::Decode(SharedCudaPtrVector<char> input)
{
	auto metadata = input[0];
	auto data = input[1];

	int size = data->size()/sizeof(T) + 1;
	auto result = CudaPtr<T>::make_shared(size);

	// Calculate deltas
	thrust::device_ptr<T> data_ptr((T*)data->get());
	thrust::device_ptr<T> result_ptr(result->get());
	thrust::inclusive_scan(data_ptr, data_ptr+(size-1), result_ptr+1);
	result_ptr[0] = 0;

	// Add first value all elements
	this->_policy.setSize(size);
	hipLaunchByPtr(this->_policy, addValueKernel<T>,
		result->get(),
		size,
		(T*)metadata->get()
	);

	return result;
}

#define DELTA_ENCODING_SPEC(X) \
	template SharedCudaPtrVector<char> DeltaEncoding::Encode<X>(SharedCudaPtr<X> data); \
	template SharedCudaPtr<X> DeltaEncoding::Decode<X>(SharedCudaPtrVector<char> data);
FOR_EACH(DELTA_ENCODING_SPEC, float, int, long long, unsigned int)


} /* namespace ddj */
