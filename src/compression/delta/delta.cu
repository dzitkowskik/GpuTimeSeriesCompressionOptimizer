#include "hip/hip_runtime.h"
#include "delta.cuh"
#include "helpers/helper_macros.h"
#include <hip/hip_runtime_api.h>

//a  a  b  b  a  a  b  b  a  a  b  b  a  a
//1  2  2  3  3  4  4  4  5  1  2  3  3  3
// 1  0  1  0  1  0  0  1  -4 1  1  0  0

#define DELTA_ENCODING_GPU_BLOCK_SIZE 64
#define DELTA_DECODING_GPU_BLOCK_SIZE 64

namespace ddj
{

template<typename T>
__global__ void deltaEncodeKernel(T* data, int size, T* result)
{
	unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx+1 >= size) return;
	register T v1 = data[idx];
	register T v2 = data[idx+1];
	result[idx] = v2 - v1;
}

template<typename T>
T* deltaEncode(T* data, int size, T& first)
{
	int result_size = size - 1;
	int block_size = DELTA_ENCODING_GPU_BLOCK_SIZE;
	int block_cnt = (size + block_size - 1) / block_size;

	T* result;
	CUDA_CALL( hipMalloc((void**)&result, result_size*sizeof(T)) );
	deltaEncodeKernel<T><<<block_size, block_cnt>>>(data, size, result);
	CUDA_CALL( hipMemcpy(&first, data, sizeof(T), hipMemcpyDeviceToHost) );
	hipDeviceSynchronize();

	return result;
}

template<typename T>
void deltaEncodeInPlace(T* data, int size)
{
	int block_size = DELTA_ENCODING_GPU_BLOCK_SIZE;
	int block_cnt = (size + block_size - 1) / block_size;

	deltaEncodeKernel<T><<<block_size, block_cnt>>>(data, size, data+1);
	hipDeviceSynchronize();
}

template<typename T>
__global__ void deltaDecodeKernel(T* data, int size, T first, T* result)
{
	unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx >= size) return;
	for(int i=0; i<idx; i++)
		first += data[i];
	result[idx] = first;
}

template<typename T>
T* deltaDecode(T* data, int size, T first)
{
	int block_size = DELTA_DECODING_GPU_BLOCK_SIZE;
	int block_cnt = (size + block_size - 1) / block_size;

	T* result;
	CUDA_CALL( hipMalloc((void**)&result, size*sizeof(T)) );
	deltaDecodeKernel<T><<<block_size, block_cnt>>>(data, size, first, result);
	hipDeviceSynchronize();

	return result;
}

template<typename T>
void deltaDecodeInPlace(T* data, int size)
{
	int block_size = DELTA_DECODING_GPU_BLOCK_SIZE;
	int block_cnt = (size + block_size - 1) / block_size;

	T first;
	T* result;
	CUDA_CALL( hipMemcpy(&first, data, sizeof(T), hipMemcpyDeviceToHost) );
	CUDA_CALL( hipMalloc((void**)result, size*sizeof(T)) );
	deltaDecodeKernel<T><<<block_size, block_cnt>>>(data+1, size, first, result);
	hipDeviceSynchronize();
	CUDA_CALL( hipMemcpy(data, result, size*sizeof(T), hipMemcpyDeviceToDevice) );
	CUDA_CALL( hipFree(result) );
}

template float* deltaEncode<float>(float* data, int size, float& first);
template void deltaEncodeInPlace<float>(float* data, int size);
template float* deltaDecode<float>(float* data, int size, float first);
template void deltaDecodeInPlace<float>(float* data, int size);

} /* namespace ddj */
