#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/reduce.h>
#include "thrust_rle.cuh"

#include <iostream>
#include <iterator>

namespace ddj {

void* ThrustRleCompression::Encode(void* data, const int in_size, int& out_size)
{
    thrust::device_ptr<float> d_ptr(reinterpret_cast<float*>(data));
    thrust::device_vector<float> input(d_ptr, d_ptr + in_size);
    thrust::device_vector<float> output(in_size);
    thrust::device_vector<int>  lengths(in_size);

    #if DDJ_THRUST_RLE_DEBUG
        // print the initial data
        std::cout << "input data:" << std::endl;
        thrust::copy(input.begin(), input.end(), std::ostream_iterator<float>(std::cout, ""));
        std::cout << std::endl << std::endl;
    #endif

    // compute run lengths
    auto result = thrust::reduce_by_key(
        input.begin(),
        input.end(),
        thrust::constant_iterator<int>(1),
        output.begin(),
        lengths.begin());

    size_t len = result.first - output.begin();

    #if DDJ_THRUST_RLE_DEBUG
        // print the output
        std::cout << "run-length encoded output:" << std::endl;
        for(size_t i = 0; i < len; i++)
            std::cout << "(" << output[i] << "," << lengths[i] << ")";
        std::cout << std::endl;
    #endif

    // prepare data
    int* raw_ptr;
    hipMalloc((void **) &raw_ptr, len * sizeof(int) + len * sizeof(float));
    float* raw_ptr_2 = reinterpret_cast<float*>(raw_ptr + len);

    thrust::device_ptr<int> dev_ptr_int(raw_ptr);
    thrust::device_ptr<float> dev_ptr_float(raw_ptr_2);
    thrust::copy(lengths.begin(), lengths.end(), dev_ptr_int);
    thrust::copy(output.begin(), output.end(), dev_ptr_float);

    out_size = len;
    return raw_ptr;
}

} /* namespace ddj */
