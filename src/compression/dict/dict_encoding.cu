#include "hip/hip_runtime.h"
#include "dict_encoding.hpp"
#include "core/cuda_macros.cuh"
#include "core/cuda_launcher.cuh"

#include "util/stencil/stencil.hpp"
#include "util/histogram/histogram.hpp"
#include "compression/unique/unique_encoding.hpp"
#include "util/other/cuda_array_reduce.cuh"

#include <thrust/device_ptr.h>
#include <thrust/count.h>

namespace ddj {

template<typename T>
__global__ void getMostFrequentStencilKernel(
		T* data,
		int size,
		T* mostFrequent,
		int freqCnt,
		int* output)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx >= size) return;
    T value = data[idx];
    output[idx] = 0;
    for(int i = 0; i < freqCnt; i++)
    {
        if(value == mostFrequent[i])
        {
            output[idx] = 1;
        }
    }
}

template<typename T>
SharedCudaPtr<int> DictEncoding::GetMostFrequentStencil(
		SharedCudaPtr<T> data,
		SharedCudaPtr<T> mostFrequent)
{
    auto result = CudaPtr<int>::make_shared(data->size());

    this->_policy.setSize(data->size());
    hipLaunchByPtr(this->_policy, getMostFrequentStencilKernel<T>,
        data->get(),
        data->size(),
        mostFrequent->get(),
        mostFrequent->size(),
        result->get());

    hipDeviceSynchronize();
    return result;
}

// DICT ENCODING ALGORITHM
//
//  1. CREATE HISTOGRAM
//  2. GET N MOST FREQUENT VALUES
//  3. SPLIT TO MOST FREQUENT AND OTHERS
//  4. PACK STENCIL
//  5. COMPRESS MOST FREQUENT
//       a) GET DISTINCT NUMBERS AND GIVE THEM THE SHORTEST UNIQUE KEYS POSSIBLE
//       b) LEAVE N UNIQUE VALUES AT BEGINNING
//       c) REPLACE OTHER OCCURENCES OF THESE NUMBERS BY THEIR CODES (GREY CODE)
//  6. RETURN A VECTOR (STENCIL, MOST FREQUENT (COMPRESSED), OTHERS (UNCOMPRESSED))
template<typename T>
SharedCudaPtrVector<char> DictEncoding::Encode(SharedCudaPtr<T> data)
{
	CUDA_ASSERT_RETURN( hipGetLastError() );
    LOG4CPLUS_INFO_FMT(_logger, "DICT encoding START: data size = %lu", data->size());

	if(data->size() <= 0)
		return SharedCudaPtrVector<char>{
			CudaPtr<char>::make_shared(),
			CudaPtr<char>::make_shared(),
			CudaPtr<char>::make_shared()};

	auto mostFrequent = Histogram().GetMostFrequent(data, this->_freqCnt);
    auto mostFrequentStencil = GetMostFrequentStencil(data, mostFrequent);
    auto splittedData = this->_splitter.Split(data, mostFrequentStencil);
    auto packedMostFrequentStencil = Stencil(mostFrequentStencil).pack();
    auto mostFrequentCompressed = UniqueEncoding().CompressUnique(std::get<0>(splittedData), mostFrequent);
    auto otherData = MoveSharedCudaPtr<T, char>(std::get<1>(splittedData));

	LOG4CPLUS_TRACE_FMT(
		_logger,
		"DICT ENCODED output[0] size = %lu, output[1] size = %lu, output[2] size = %lu",
	 	packedMostFrequentStencil->size(), mostFrequentCompressed->size(), otherData->size()
	);

	CUDA_ASSERT_RETURN( hipGetLastError() );
	LOG4CPLUS_INFO(_logger, "DICT enoding END");

    return SharedCudaPtrVector<char> {packedMostFrequentStencil, mostFrequentCompressed, otherData};
}

// DICT DECODING ALGORITHM
//
// 1. UNPACK STENCIL
// 2. GET MOST FREQUENT DATA COMPRESSED AND DECOMPRESS IT
// 3. USE STENCIL TO MERGE MOST FREQUENT DATA AND OTHER
// 4. RETURN MERGED DATA
template<typename T>
SharedCudaPtr<T> DictEncoding::Decode(SharedCudaPtrVector<char> input)
{
	LOG4CPLUS_INFO_FMT(
		_logger,
		"DICT decoding START: input[0] size = %lu, input[1] size = %lu, input[2] size = %lu",
		input[0]->size(), input[1]->size(), input[2]->size()
	);

	if(input[1]->size() <= 0 && input[2]->size() <= 0)
		return CudaPtr<T>::make_shared();

	// UNPACK STENCIL
	auto stencil = Stencil(input[0]);
	auto mostFrequentCompressed = input[1];
	auto other = MoveSharedCudaPtr<char, T>(input[2]);

	// DECOMPRESS MOST FREQUENT
	auto mostFrequent = UniqueEncoding().DecompressUnique<T>(mostFrequentCompressed);

	// MERGE DATA
	auto result = this->_splitter.Merge<T>(std::make_tuple(mostFrequent, other), *stencil);

	CUDA_ASSERT_RETURN( hipGetLastError() );
    LOG4CPLUS_INFO(_logger, "DICT decoding END");

	return result;
}

size_t DictEncoding::GetCompressedSize(SharedCudaPtr<char> data, DataType type)
{
	if(data->size() <= 0) return 0;
	switch(type)
	{
		case DataType::d_int:
			return GetCompressedSize(boost::reinterpret_pointer_cast<CudaPtr<int>>(data));
		case DataType::d_float:
			return GetCompressedSize(boost::reinterpret_pointer_cast<CudaPtr<float>>(data));
		default:
			throw NotImplementedException("No DictEncoding::GetCompressedSize implementation for that type");
	}
}

template<typename T>
size_t DictEncoding::GetCompressedSize(SharedCudaPtr<T> data)
{
	if(data->size() <= 0) return 0;
	auto mostFrequent = Histogram().GetMostFrequent(data, this->_freqCnt);
	int freqCnt = mostFrequent->size();
	auto mostFrequentStencil = GetMostFrequentStencil(data, mostFrequent);
	size_t mostFrequentCompressedSize = 2*sizeof(size_t) + freqCnt*sizeof(T);
	int dataPerOutputCnt = (8 * sizeof(unsigned int)) / ALT_BITLEN(freqCnt - 1);
	int stencilDataCnt = reduce_thrust(mostFrequentStencil, thrust::plus<int>());
	int outputSize = (stencilDataCnt + dataPerOutputCnt - 1) / dataPerOutputCnt;
	mostFrequentCompressedSize += outputSize * sizeof(unsigned int);
	int othersCnt = data->size() - stencilDataCnt;
	size_t otherDataSize = othersCnt * sizeof(T);
	return mostFrequentCompressedSize + otherDataSize;
}

#define DICT_ENCODING_SPEC(X) \
	template SharedCudaPtrVector<char> DictEncoding::Encode<X>(SharedCudaPtr<X> data); \
	template SharedCudaPtr<X> DictEncoding::Decode<X>(SharedCudaPtrVector<char> data);
FOR_EACH(DICT_ENCODING_SPEC, short, char, double, float, int, long, long long, unsigned int)

} /* namespace ddj */
