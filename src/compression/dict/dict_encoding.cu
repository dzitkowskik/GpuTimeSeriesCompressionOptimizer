#include "hip/hip_runtime.h"
#include "dict_encoding.hpp"
#include "core/cuda_macros.cuh"
#include "core/cuda_launcher.cuh"
#include "helpers/helper_print.hpp"
#include "util/stencil/stencil.hpp"
#include "util/histogram/histogram.hpp"
#include "compression/unique/unique_encoding.hpp"
#include "util/other/cuda_array_reduce.cuh"

#include <thrust/device_ptr.h>
#include <thrust/count.h>

namespace ddj {

template<typename T>
__global__ void getMostFrequentStencilKernel(
		T* data,
		int size,
		T* mostFrequent,
		int freqCnt,
		int* output)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx >= size) return;
    T value = data[idx];
    output[idx] = 0;
    for(int i = 0; i < freqCnt; i++)
    {
        if(value == mostFrequent[i])
        {
            output[idx] = 1;
        }
    }
}

template<typename T>
SharedCudaPtr<int> DictEncoding::GetMostFrequentStencil(
		SharedCudaPtr<T> data,
		SharedCudaPtr<T> mostFrequent)
{
    auto result = CudaPtr<int>::make_shared(data->size());

    this->_policy.setSize(data->size());
    hipLaunchByPtr(this->_policy, getMostFrequentStencilKernel<T>,
        data->get(),
        data->size(),
        mostFrequent->get(),
        mostFrequent->size(),
        result->get());

    hipDeviceSynchronize();
    return result;
}

// DICT ENCODING ALGORITHM
//
//  1. CREATE HISTOGRAM
//  2. GET N MOST FREQUENT VALUES
//  3. SPLIT TO MOST FREQUENT AND OTHERS
//  4. PACK STENCIL
//  5. COMPRESS MOST FREQUENT
//       a) GET DISTINCT NUMBERS AND GIVE THEM THE SHORTEST UNIQUE KEYS POSSIBLE
//       b) LEAVE N UNIQUE VALUES AT BEGINNING
//       c) REPLACE OTHER OCCURENCES OF THESE NUMBERS BY THEIR CODES (GREY CODE)
//  6. RETURN A VECTOR (STENCIL, MOST FREQUENT (COMPRESSED), OTHERS (UNCOMPRESSED))
template<typename T>
SharedCudaPtrVector<char> DictEncoding::Encode(SharedCudaPtr<T> data)
{
	auto mostFrequent = Histogram().GetMostFrequent(data, this->_freqCnt);
    auto mostFrequentStencil = GetMostFrequentStencil(data, mostFrequent);
    auto splittedData = this->_splitter.Split(data, mostFrequentStencil);
    auto packedMostFrequentStencil = Stencil(mostFrequentStencil).pack();
    auto mostFrequentCompressed = UniqueEncoding().CompressUnique(std::get<0>(splittedData), mostFrequent);
    auto otherData = MoveSharedCudaPtr<T, char>(std::get<1>(splittedData));
    return SharedCudaPtrVector<char> {packedMostFrequentStencil, mostFrequentCompressed, otherData};
}

// DICT DECODING ALGORITHM
//
// 1. UNPACK STENCIL
// 2. GET MOST FREQUENT DATA COMPRESSED AND DECOMPRESS IT
// 3. USE STENCIL TO MERGE MOST FREQUENT DATA AND OTHER
// 4. RETURN MERGED DATA
template<typename T>
SharedCudaPtr<T> DictEncoding::Decode(SharedCudaPtrVector<char> input)
{
	// UNPACK STENCIL
	auto stencil = Stencil(input[0]);
	auto mostFrequentCompressed = input[1];
	auto other = MoveSharedCudaPtr<char, T>(input[2]);

	// DECOMPRESS MOST FREQUENT
	auto mostFrequent = UniqueEncoding().DecompressUnique<T>(mostFrequentCompressed);

	// MERGE DATA
	return this->_splitter.Merge<T>(std::make_tuple(mostFrequent, other), *stencil);
}

size_t DictEncoding::GetCompressedSize(SharedCudaPtr<char> data, DataType type)
{
	if(data->size() <= 0) return 0;
	switch(type)
	{
		case DataType::d_int:
			return GetCompressedSize(boost::reinterpret_pointer_cast<CudaPtr<int>>(data));
		case DataType::d_float:
			return GetCompressedSize(boost::reinterpret_pointer_cast<CudaPtr<float>>(data));
		default:
			throw NotImplementedException("No DictEncoding::GetCompressedSize implementation for that type");
	}
}

template<typename T>
size_t DictEncoding::GetCompressedSize(SharedCudaPtr<T> data)
{
	if(data->size() <= 0) return 0;
	auto mostFrequent = Histogram().GetMostFrequent(data, this->_freqCnt);
	int freqCnt = mostFrequent->size();
	auto mostFrequentStencil = GetMostFrequentStencil(data, mostFrequent);
	size_t mostFrequentCompressedSize = 2*sizeof(size_t) + freqCnt*sizeof(T);
	int dataPerOutputCnt = (8 * sizeof(unsigned int)) / ALT_BITLEN(freqCnt - 1);
	int stencilDataCnt = reduce_thrust(mostFrequentStencil, thrust::plus<int>());
	int outputSize = (stencilDataCnt + dataPerOutputCnt - 1) / dataPerOutputCnt;
	mostFrequentCompressedSize += outputSize * sizeof(unsigned int);
	int othersCnt = data->size() - stencilDataCnt;
	size_t otherDataSize = othersCnt * sizeof(T);
	return mostFrequentCompressedSize + otherDataSize;
}

#define DICT_ENCODING_SPEC(X) \
	template SharedCudaPtrVector<char> DictEncoding::Encode<X>(SharedCudaPtr<X> data); \
	template SharedCudaPtr<X> DictEncoding::Decode<X>(SharedCudaPtrVector<char> data);
FOR_EACH(DICT_ENCODING_SPEC, float, int, long, long long, unsigned int)

} /* namespace ddj */
