#include "hip/hip_runtime.h"
/*
 *  cuda_array_statistics.cu
 *
 *  Created on: 21-10-2015
 *      Author: Karol Dzitkowski
 */

#include "util/statistics/cuda_array_statistics.hpp"
#include "core/cuda_macros.cuh"
#include <thrust/device_ptr.h>
#include <thrust/extrema.h>

namespace ddj {

template<typename T> std::tuple<T,T> CudaArrayStatistics::MinMax(SharedCudaPtr<T> data)
{
	thrust::device_ptr<T> dp(data->get());
	auto tuple = thrust::minmax_element(dp, dp+data->size());
	T min = *(tuple.first);
	T max = *(tuple.second);
	return std::make_tuple(min, max);
}

template<typename T>
char CudaArrayStatistics::MinBitCnt(SharedCudaPtr<T> data)
{
	auto minMax = MinMax(data);
	int result = 32;
	if (std::get<0>(minMax) >= 0)
		result = ALT_BITLEN(std::get<1>(minMax));
	return result;
}

#define CUDA_ARRAY_STATISTICS_SPEC(X) \
	template std::tuple<X,X> CudaArrayStatistics::MinMax<X>(SharedCudaPtr<X>); \
	template char CudaArrayStatistics::MinBitCnt<X>(SharedCudaPtr<X>);
FOR_EACH(CUDA_ARRAY_STATISTICS_SPEC, float, int, long long, unsigned int)

} /* namespace ddj */
