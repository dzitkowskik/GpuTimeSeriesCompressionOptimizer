#include "hip/hip_runtime.h"
#include "cuda_array_generator.hpp"
#include "helpers/helper_cuda.cuh"

namespace ddj {

template<typename T>
__global__ void createConsecutiveNumbersArrayKernel(T* data, int size, T start)
{
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= size) return;
	data[idx] = start + idx;
}

template<typename T>
__global__ void createConsecutiveNumbersArrayWithStepKernel(T* data, int size, T start, T step)
{
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= size) return;
	data[idx] = start + (idx * step);
}

template<typename T> SharedCudaPtr<T>
CudaArrayGenerator::CreateConsecutiveNumbersArray(int size, T start)
{
    auto result = CudaPtr<T>::make_shared(size);

    this->_policy.setSize(size);
    hipLaunchByPtr(this->_policy, createConsecutiveNumbersArrayKernel<T>,
        result->get(),
        size,
        start
    );

    hipDeviceSynchronize();
    return result;
}

template<typename T> SharedCudaPtr<T>
CudaArrayGenerator::CreateConsecutiveNumbersArray(int size, T start, T step)
{
    auto result = CudaPtr<T>::make_shared(size);

    this->_policy.setSize(size);
    hipLaunchByPtr(this->_policy, createConsecutiveNumbersArrayWithStepKernel<T>,
        result->get(),
        size,
        start,
        step
    );

    hipDeviceSynchronize();
    return result;
}

#define CUDA_ARRAY_GENERATOR_SPEC(X) \
	template SharedCudaPtr<X> CudaArrayGenerator::CreateConsecutiveNumbersArray<X>(int, X); \
	template SharedCudaPtr<X> CudaArrayGenerator::CreateConsecutiveNumbersArray<X>(int, X, X);
FOR_EACH(CUDA_ARRAY_GENERATOR_SPEC, float, int, long long, unsigned int)

} /* namespace ddj */
