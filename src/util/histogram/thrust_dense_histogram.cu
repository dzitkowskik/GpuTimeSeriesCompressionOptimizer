#include "hip/hip_runtime.h"
#include "util/histogram/histogram.hpp"
#include "core/macros.h"

#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/binary_search.h>
#include <thrust/adjacent_difference.h>
#include <thrust/iterator/counting_iterator.h>

namespace ddj {

template<typename T>
SharedCudaPtrPair<T, int> Histogram::ThrustDenseHistogram(SharedCudaPtr<T> data)
{
    thrust::device_ptr<T> data_ptr(data->get());
    thrust::device_vector<T> input_keys_dvec(data_ptr, data_ptr+data->size());

    thrust::sort(input_keys_dvec.begin(), input_keys_dvec.end());

    // number of histogram bins is equal to the maximum value plus one
    int num_bins = input_keys_dvec.back() - input_keys_dvec.front() + 1;

    // allocate histogram storage
    auto output_keys = CudaPtr<T>::make_shared(num_bins);
    auto output_counts = CudaPtr<int>::make_shared(num_bins);
    thrust::device_ptr<T> output_keys_ptr(output_keys->get());
    thrust::device_ptr<int> output_counts_ptr(output_counts->get());
    thrust::device_vector<int> output_counts_vec(num_bins);

    // find the end of each bin of values (cumulative histogram)
    thrust::counting_iterator<T> search_begin(input_keys_dvec.front());
    thrust::upper_bound(input_keys_dvec.begin(),
    					input_keys_dvec.end(),
						search_begin,
						search_begin + num_bins,
						output_counts_vec.begin());

    // compute the histogram by taking differences of the cumulative histogram
    thrust::adjacent_difference(output_counts_vec.begin(), output_counts_vec.end(), output_counts_vec.begin());

    // keys are sequence from input keys min to max
    thrust::counting_iterator<T> keys_begin(input_keys_dvec.front());
    thrust::copy(keys_begin, keys_begin+num_bins, output_keys_ptr);
    thrust::copy(output_counts_vec.begin(), output_counts_vec.end(), output_counts_ptr);

    return SharedCudaPtrPair<T, int>(output_keys, output_counts);
}

#define THRUST_DENSE_HISTOGRAM_SPEC(X) \
	template SharedCudaPtrPair<X, int> Histogram::ThrustDenseHistogram<X>(SharedCudaPtr<X>);
FOR_EACH(THRUST_DENSE_HISTOGRAM_SPEC, char, short, double, float, int, long, long long, unsigned int)

} /* namespace ddj */
